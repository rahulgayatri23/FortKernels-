#include "hip/hip_runtime.h"
#include "CustomComplex.h"

// Atomic add operation for double
#if defined( __CUDA_ARCH__ ) && __CUDA_ARCH__ >= 600
#define atomicAdd2 atomicAdd
#else
__device__ double atomicAdd2( double *address, double val )
{
    unsigned long long int *address_as_ull = (unsigned long long int *) address;
    unsigned long long int old             = *address_as_ull, assumed;
    do {
        assumed = old;
        old     = atomicCAS( address_as_ull, assumed,
            __double_as_longlong( val + __longlong_as_double( assumed ) ) );
    } while ( assumed != old );
    return __longlong_as_double( old );
}
#endif

__device__ void d_compute_fact(double wx, int nFreq, double *dFreqGrid, double &fact1, double &fact2, int &ifreq, int loop, bool flag_occ)
{
    if(loop == 1 && wx > 0.00)
    {
            for(int ijk = 0; ijk < nFreq-1; ++ijk)
            {
                if(wx > dFreqGrid[ijk] && wx < dFreqGrid[ijk+1])
                ifreq = ijk;
            }
            if(ifreq == 0) ifreq = nFreq-2;
            fact1 = (dFreqGrid[ifreq+1] - wx) / (dFreqGrid[ifreq+1] - dFreqGrid[ifreq]);
            fact2 = (wx - dFreqGrid[ifreq]) / (dFreqGrid[ifreq+1] - dFreqGrid[ifreq]);
    }
    else if(loop == 1)
    {
        for(int ijk = 0; ijk < nFreq-1; ++ijk)
        {
            if(-wx > dFreqGrid[ijk] && -wx < dFreqGrid[ijk+1])
                ifreq = ijk;
        }
        if(ifreq == 0) ifreq = nFreq-2;
        fact1 = (dFreqGrid[ifreq+1] + wx) / (dFreqGrid[ifreq+1] - dFreqGrid[ifreq]);
        fact2 = (-dFreqGrid[ifreq] - wx) / (dFreqGrid[ifreq+1] - dFreqGrid[ifreq]);
    }
    if(loop == 2 && wx > 0.00)
    {
            for(int ijk = 0; ijk < nFreq-1; ++ijk)
            {
                if(wx > dFreqGrid[ijk] && wx < dFreqGrid[ijk+1])
                    ifreq = ijk;
            }
            if(ifreq == -1) ifreq = nFreq-2;
            fact1 = -0.5 * (dFreqGrid[ifreq+1] - wx) / (dFreqGrid[ifreq+1] - dFreqGrid[ifreq]); 
            fact2 = -0.5 * (wx - dFreqGrid[ifreq]) / (dFreqGrid[ifreq+1] - dFreqGrid[ifreq]); 
    }
    else if(loop == 2 && flag_occ)
    {
        wx = -wx; ifreq = 0;
        for(int ijk = 0; ijk < nFreq-1; ++ijk)
        {
            if(wx > dFreqGrid[ijk] && wx < dFreqGrid[ijk+1])
                ifreq = ijk;
        }
        if(ifreq == 0) ifreq = nFreq-2;
        fact1 = (dFreqGrid[ifreq+1] - wx) / (dFreqGrid[ifreq+1] - dFreqGrid[ifreq]); 
        fact2 = (wx - dFreqGrid[ifreq]) / (dFreqGrid[ifreq+1] - dFreqGrid[ifreq]); 

    }
}

__device__ void d_ssxDittt_kernel(int *inv_igp_index, int *indinv, CustomComplex *aqsmtemp, CustomComplex *aqsntemp, double *vcoul, CustomComplex *I_eps_array, CustomComplex &ssxDittt, int ngpown, int ncouls, int n1,int ifreq, double fact1, double fact2)
{
    double ssxDittt_re = 0.00, ssxDittt_im = 0.00;
    for(int my_igp = 0; my_igp < ngpown; ++my_igp)
    {
        int indigp = inv_igp_index[my_igp];
        int igp = indinv[indigp];
        CustomComplex ssxDit(0.00, 0.00);
        CustomComplex ssxDitt(0.00, 0.00);

        for(int ig = 0; ig < ncouls; ++ig)
        {
            ssxDit = I_eps_array[ifreq*ngpown*ncouls + my_igp*ncouls + ig] * fact1 + \
                                         I_eps_array[(ifreq+1)*ngpown*ncouls + my_igp*ncouls + ig] * fact2;

            ssxDitt += aqsntemp[n1*ncouls + ig] * CustomComplex_conj(aqsmtemp[n1*ncouls + igp]) * ssxDit * vcoul[igp];
        }
        ssxDittt_re += CustomComplex_real(ssxDitt);
        ssxDittt_im += CustomComplex_imag(ssxDitt);
    }
    ssxDittt = CustomComplex (ssxDittt_re, ssxDittt_im);
}

__device__ void d_schDttt_corKernel1(CustomComplex &schDttt_cor, int *inv_igp_index, int *indinv, CustomComplex *I_epsR_array, CustomComplex *I_epsA_array, CustomComplex *aqsmtemp, CustomComplex *aqsntemp, double *vcoul, int ncouls, int ifreq, int ngpown, int n1, double fact1, double fact2)
{
    int blkSize = 512;
    double schDttt_cor_re = 0.00, schDttt_cor_im = 0.00, \
        schDttt_re = 0.00, schDttt_im = 0.00;
    for(int igbeg = 0; igbeg < ncouls; igbeg += blkSize)
    {
        for(int my_igp = 0; my_igp < ngpown; ++my_igp)
        {
            for(int ig = igbeg; ig < min(ncouls, igbeg+blkSize); ++ig)
            {
                int indigp = inv_igp_index[my_igp] ;
                int igp = indinv[indigp];
                CustomComplex sch2Dt = (I_epsR_array[ifreq*ngpown*ncouls + my_igp*ncouls + ig] - I_epsA_array[ifreq*ngpown*ncouls + my_igp*ncouls + ig]) * fact1 + \
                                            (I_epsR_array[(ifreq+1)*ngpown*ncouls + my_igp*ncouls + ig] - I_epsA_array[(ifreq+1)*ngpown*ncouls + my_igp*ncouls + ig]) * fact2;
                CustomComplex sch2Dtt = aqsntemp[n1*ncouls + ig] * CustomComplex_conj(aqsmtemp[n1*ncouls + igp]) * sch2Dt * vcoul[igp];


                schDttt_re += CustomComplex_real(sch2Dtt) ;
                schDttt_im += CustomComplex_imag(sch2Dtt) ;
                schDttt_cor_re += CustomComplex_real(sch2Dtt) ;
                schDttt_cor_im += CustomComplex_imag(sch2Dtt) ;
            }
        }
    }
    schDttt_cor = CustomComplex (schDttt_cor_re, schDttt_cor_im);
    printf("From schDttt_corKernel1, schDttt_cor = \n");
    schDttt_cor.print();

}


__device__ void d_schDttt_corKernel2(CustomComplex &schDttt_cor, int *inv_igp_index, int *indinv, CustomComplex *I_epsR_array, CustomComplex *I_epsA_array, CustomComplex *aqsmtemp, CustomComplex *aqsntemp, double *vcoul, int ncouls, int ifreq, int ngpown, int n1, double fact1, double fact2)
{
    double schDttt_cor_re = 0.00, schDttt_cor_im = 0.00;
    for(int my_igp = 0; my_igp < ngpown; ++my_igp)
    {
        for(int ig = 0; ig < ncouls; ++ig)
        {
            int indigp = inv_igp_index[my_igp] ;
            int igp = indinv[indigp];
            CustomComplex sch2Dt = ((I_epsR_array[ifreq*ngpown*ncouls + my_igp*ncouls + ig] - I_epsA_array[ifreq*ncouls*ngpown + my_igp*ncouls + ig]) * fact1 + \
                                        (I_epsR_array[(ifreq+1)*ngpown*ncouls + my_igp*ncouls + ig] - I_epsA_array[(ifreq+1)*ngpown*ncouls + my_igp*ncouls + ig]) * fact2) * -0.5;
            CustomComplex sch2Dtt = aqsntemp[n1*ncouls + ig] * CustomComplex_conj(aqsmtemp[n1*ncouls + igp]) * sch2Dt * vcoul[igp];
            schDttt_cor_re += CustomComplex_real(sch2Dtt) ;
            schDttt_cor_im += CustomComplex_imag(sch2Dtt) ;
        }
    }
    schDttt_cor = CustomComplex (schDttt_cor_re, schDttt_cor_im);
}

__global__ void achsDtemp_solver(int number_bands, int ngpown, int ncouls, int *inv_igp_index, int *indinv, CustomComplex *aqsntemp, CustomComplex *aqsmtemp, CustomComplex *I_epsR_array, double *vcoul, double *achsDtemp_re, double *achsDtemp_im, int numThreadsPerBlock)
{
    int n1 = blockIdx.x;
    int my_igp = blockIdx.y;
    int loopOverncouls=1, leftOverncouls=0;
    if(ncouls > numThreadsPerBlock)
    {
        loopOverncouls = ncouls / numThreadsPerBlock;
        leftOverncouls = ncouls % numThreadsPerBlock;
    }

    if( n1 < number_bands && my_igp < ngpown)
    {
        int indigp = inv_igp_index[my_igp];
        int igp = indinv[indigp];
        CustomComplex schsDtemp(0.00, 0.00);

        for( int x = 0; x < loopOverncouls && threadIdx.x < numThreadsPerBlock ; ++x)
        { 
            int ig = x*numThreadsPerBlock + threadIdx.x;
            schsDtemp = schsDtemp - aqsntemp[n1*ncouls + ig] * CustomComplex_conj(aqsmtemp[n1*ncouls + igp]) * I_epsR_array[1*ngpown*ncouls + my_igp*ncouls + ig]* vcoul[ig] * 0.5;
        }
        if(leftOverncouls)
        {
            int ig = loopOverncouls*numThreadsPerBlock + threadIdx.x;
            schsDtemp = schsDtemp - aqsntemp[n1*ncouls + ig] * CustomComplex_conj(aqsmtemp[n1*ncouls + igp]) * I_epsR_array[1*ngpown*ncouls + my_igp*ncouls + ig]* vcoul[ig] * 0.5;
        }

        atomicAdd(achsDtemp_re, CustomComplex_real(schsDtemp));
        atomicAdd(achsDtemp_im, CustomComplex_imag(schsDtemp));
    }
}

__global__ void asxDtemp_solver(int nvband, int nfreqeval, int ncouls, int ngpown, int nFreq, double freqevalmin, double freqevalstep, double occ, double *ekq, double *dFreqGrid, int *inv_igp_index, int *indinv, CustomComplex *aqsmtemp, CustomComplex *aqsntemp, double *vcoul, CustomComplex *I_epsR_array, CustomComplex *I_epsA_array, double *asxDtemp_re, double *asxDtemp_im)
{
    CustomComplex ssxDittt(0.00, 0.00);
    int n1 = blockIdx.x;
    int iw = blockIdx.y;
    if(n1 < nvband && iw < nfreqeval)
    {
        double wx = freqevalmin - ekq[n1] + freqevalstep;
        double fact1 = 0.00, fact2 = 0.00;
        int ifreq = 0;
        CustomComplex ssxDittt(0.00, 0.00);

        d_compute_fact(wx, nFreq, dFreqGrid, fact1, fact2, ifreq, 1, 0);

        if(wx > 0)
            d_ssxDittt_kernel(inv_igp_index, indinv, aqsmtemp, aqsntemp, vcoul, I_epsR_array, ssxDittt, ngpown, ncouls, n1, ifreq, fact1, fact2);
        else
            d_ssxDittt_kernel(inv_igp_index, indinv, aqsmtemp, aqsntemp, vcoul, I_epsA_array, ssxDittt, ngpown, ncouls, n1, ifreq, fact1, fact2);

        atomicAdd(&asxDtemp_re[iw], CustomComplex_real(ssxDittt * occ));
        atomicAdd(&asxDtemp_im[iw], CustomComplex_imag(ssxDittt * occ));
    }
}


__global__ void achDtemp_cor_solver(int number_bands, int nvband, int nfreqeval, int ncouls, int ngpown, int nFreq, double freqevalmin, double freqevalstep, double *ekq, double *dFreqGrid, int *inv_igp_index, int *indinv, CustomComplex *aqsmtemp, CustomComplex *aqsntemp, double *vcoul, CustomComplex *I_epsR_array, CustomComplex *I_epsA_array, CustomComplex *ach2Dtemp, double *achDtemp_cor_re, double *achDtemp_cor_im, CustomComplex *achDtemp_corb, int numThreadsPerBlock)
{
    bool flag_occ;
    int n1 = blockIdx.x;
    if(n1 < number_bands)
    {
        flag_occ = n1 < nvband;

        for(int iw = 0; iw < nfreqeval; ++iw)
        {
            CustomComplex schDi_cor(0.00, 0.00);
            CustomComplex schDi_corb(0.00, 0.00);
            double wx = freqevalmin - ekq[n1] + freqevalstep;

            double fact1 = 0.00, fact2 = 0.00;
            int ifreq = 0.00;

            d_compute_fact(wx, nFreq, dFreqGrid, fact1, fact2, ifreq, 2, flag_occ);

            if(wx > 0)
            {
                if(!flag_occ)
                    d_schDttt_corKernel1(schDi_cor, inv_igp_index, indinv, I_epsR_array, I_epsA_array, aqsmtemp, aqsntemp, vcoul,  ncouls, ifreq, ngpown, n1, fact1, fact2);
            }
            else if(flag_occ)
                d_schDttt_corKernel2(schDi_cor, inv_igp_index, indinv, I_epsR_array, I_epsA_array, aqsmtemp, aqsntemp, vcoul,  ncouls, ifreq, ngpown, n1, fact1, fact2);


//Summing up at the end of iw loop
//            ach2Dtemp[iw] += sch2Di[iw];
//            achDtemp_corb[iw] += schDi_corb[iw];
            atomicAdd2(&achDtemp_cor_re[iw], CustomComplex_real(schDi_cor));
            atomicAdd2(&achDtemp_cor_im[iw], CustomComplex_imag(schDi_cor));
//            achDtemp_cor_re[iw] += CustomComplex_real(schDi_cor);
//            achDtemp_cor_im[iw] += CustomComplex_imag(schDi_cor);

        }// iw
    } //n1
}


void d_achsDtemp_Kernel(int number_bands, int ngpown, int ncouls, int *inv_igp_index, int *indinv, CustomComplex *aqsntemp, CustomComplex *aqsmtemp, CustomComplex *I_epsR_array, double *vcoul, double *achsDtemp_re, double *achsDtemp_im)
{
    dim3 numBlocks(number_bands, ngpown);
    int numThreadsPerBlock=32;

    achsDtemp_solver<<<numBlocks, numThreadsPerBlock>>>(number_bands, ngpown, ncouls, inv_igp_index, indinv, aqsntemp, aqsmtemp, I_epsR_array, vcoul, achsDtemp_re, achsDtemp_im, numThreadsPerBlock); 
}

void d_asxDtemp_Kernel(int nvband, int nfreqeval, int ncouls, int ngpown, int nFreq, double freqevalmin, double freqevalstep, double occ, double *ekq, double *dFreqGrid, int *inv_igp_index, int *indinv, CustomComplex *aqsmtemp, CustomComplex *aqsntemp, double *vcoul, CustomComplex *I_epsR_array, CustomComplex *I_epsA_array, double *asxDtemp_re, double *asxDtemp_im)
{
    dim3 numBlocks(nvband, nfreqeval);
    int numThreadsPerBlock=1;

    asxDtemp_solver<<<numBlocks, numThreadsPerBlock>>>(nvband, nfreqeval, ncouls, ngpown, nFreq, freqevalmin, freqevalstep, occ, ekq, dFreqGrid, inv_igp_index, indinv, aqsmtemp, aqsntemp, vcoul, I_epsR_array, I_epsA_array, asxDtemp_re, asxDtemp_im);

}

void d_achDtemp_cor_Kernel(int number_bands, int nvband, int nfreqeval, int ncouls, int ngpown, int nFreq, double freqevalmin, double freqevalstep, double *ekq, double *dFreqGrid, int *inv_igp_index, int *indinv, CustomComplex *aqsmtemp, CustomComplex *aqsntemp, double *vcoul, CustomComplex *I_epsR_array, CustomComplex *I_epsA_array, CustomComplex *ach2Dtemp, double *achDtemp_cor_re, double *achDtemp_cor_im, CustomComplex *achDtemp_corb)
{
    dim3 numBlocks = number_bands;;
    int numThreadsPerBlock=1;

    achDtemp_cor_solver<<<numBlocks, numThreadsPerBlock>>>(number_bands, nvband, nfreqeval, ncouls, ngpown, nFreq, freqevalmin, freqevalstep, ekq, dFreqGrid, inv_igp_index, indinv, aqsmtemp, aqsntemp, vcoul, I_epsR_array, I_epsA_array, ach2Dtemp, achDtemp_cor_re, achDtemp_cor_im, achDtemp_corb, numThreadsPerBlock);
}
