#include "hip/hip_runtime.h"
#include "testMemBounds.h"

/*
 * Return the product of 2 complex numbers 
 */
__device__ const inline GPUComplex d_GPUComplex_product(const GPUComplex& a, const GPUComplex& b) {
    return GPUComplex(a.x * b.x - a.y*b.y, a.x * b.y + a.y*b.x);
}

__device__ void inline d_GPUComplex_Equals( GPUComplex& a, const GPUComplex & b) {
    a.x = b.x;
    a.y = b.y;
}

__device__ inline double d_GPUComplex_real( const GPUComplex& src) {
    return src.x;
}

__device__ inline double d_GPUComplex_imag( const GPUComplex& src) {
    return src.y;
}

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
        file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
        file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
    return;
}

__global__ void d_testKernel(double *achsDtemp_re, double*achsDtemp_im, GPUComplex *aqsmtemp, GPUComplex *aqsntemp)
{
    int n1 = blockIdx.x;


    if(n1 == 0)
        printf("From testKernel N = %d\t M = %d\t \n", N, M);

    if(n1 < N)
    {
        double achsDtemp_re_loc = 0.00, achsDtemp_im_loc = 0.00;
        int threadId = threadIdx.x;
        if(threadId == 0)
        {
            for(int ig = 0; ig < M; ++ig)
            {
                achsDtemp_re_loc += d_GPUComplex_real(d_GPUComplex_product(aqsmtemp[n1*M + ig] , aqsntemp[n1*M + ig]));
                achsDtemp_im_loc += d_GPUComplex_imag(d_GPUComplex_product(aqsmtemp[n1*M + ig] , aqsntemp[n1*M + ig]));
            }
            atomicAdd(achsDtemp_re, achsDtemp_re_loc);
            atomicAdd(achsDtemp_im, achsDtemp_im_loc);
        }
    }
}

void testMemBounds_cuKernel(GPUComplex &achsDtemp, GPUComplex *aqsmtemp, GPUComplex *aqsntemp)
{
    GPUComplex *d_aqsmtemp, *d_aqsntemp;
    double achsDtemp_re = 0.00, achsDtemp_im = 0.00;
    double *d_achsDtemp_re , *d_achsDtemp_im;

    CudaSafeCall(hipMallocManaged((void**) &d_aqsmtemp, N * M *sizeof(GPUComplex)));
    CudaSafeCall(hipMallocManaged((void**) &d_aqsntemp, N * M *sizeof(GPUComplex)));
    CudaSafeCall(hipMallocManaged((void**) &d_achsDtemp_re, sizeof(double)));
    CudaSafeCall(hipMallocManaged((void**) &d_achsDtemp_im, sizeof(double)));

    CudaSafeCall(hipMemcpy(d_aqsmtemp, aqsmtemp, N*M*sizeof(GPUComplex), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_aqsntemp, aqsntemp, N*M*sizeof(GPUComplex), hipMemcpyHostToDevice));


    d_testKernel<<<N, 1>>> (d_achsDtemp_re, d_achsDtemp_im, d_aqsmtemp, d_aqsntemp);

    CudaSafeCall(hipMemcpy(&achsDtemp_re, d_achsDtemp_re, sizeof(double), hipMemcpyDeviceToHost));
    CudaSafeCall(hipMemcpy(&achsDtemp_im, d_achsDtemp_im, sizeof(double), hipMemcpyDeviceToHost));

    GPUComplex tmp(achsDtemp_re, achsDtemp_im);
    achsDtemp = tmp;

    hipFree(d_aqsmtemp);
    hipFree(d_aqsntemp);
    hipFree(d_achsDtemp_re);
    hipFree(d_achsDtemp_im);
}
